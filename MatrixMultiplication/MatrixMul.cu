#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


#include ""

#include <time.h>
#include <chrono> 
#include <iostream> 
using namespace std;
using namespace std::chrono;

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A,
                                                        float *B, int wA,
                                                        int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
            a <= aEnd;
            a += aStep, b += bStep) {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void ConstantInit(float *data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

int matrixMultiplyWithoutCUDA(float* C, float* A, float* B, const dim3& dimsA, const dim3& dimsB) {
    //printf("Row of A: %d, Column of A: %d\n", dimsA.y, dimsA.x);
    //printf("Row of B: %d, Column of B: %d\n", dimsB.y, dimsB.x);

    int x = dimsA.y; // row of A
    int y = dimsA.x; // column of A and row of B
    int z = dimsB.x; // column of B
                     // So, row and column of C is x & z, respectively

    //x = 6;
    //y = 4;
    //z = 5;

    for (int i = 0; i < x; i++) {
        for (int j = 0; j < z; j++) {
            int sum = 0;
            for (int k = 0; k < y; k++) {
                int actualIndexOfA = i * y + k;
                int actualIndexOfB = z * k + j;
                //printf("(A: %d, B: %d), ", actualIndexOfA, actualIndexOfB);
                sum = sum + (A[actualIndexOfA] * B[actualIndexOfB]);
            }
            int actualIndexOfC = i * z + j;
            //printf("(Stored at C: %d\n", actualIndexOfC);
            C[actualIndexOfC] = sum;
        }
    }
    //printf("End of Without CUDA\n");
    return 0;
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));

    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    //printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16) {
        MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    } else {
        MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    }

    //printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    //hipEvent_t start;
    //checkCudaErrors(hipEventCreate(&start));

    //hipEvent_t stop;
    //checkCudaErrors(hipEventCreate(&stop));

    // Record the start event
    //checkCudaErrors(hipEventRecord(start, NULL));

    // Execute the kernel
    int nIter = 1; // was 300

    auto start1 = high_resolution_clock::now();
    
    for (int j = 0; j < nIter; j++) {
        if (block_size == 16) {
            MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        } else {
            MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        }
    }

    hipDeviceSynchronize();

    auto stop1 = high_resolution_clock::now();

    auto duration1 = duration_cast<microseconds>(stop1 - start1);

    cout << duration1.count() << ", ";

    // Record the stop event
    //checkCudaErrors(hipEventRecord(stop, NULL));

    // Wait for the stop event to complete
    //checkCudaErrors(hipEventSynchronize(stop));

    //float msecTotal = 0.0f;
    //checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    //float msecPerMatrixMul = msecTotal / nIter;
    //double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
    //                           static_cast<double>(dimsA.y) *
    //                           static_cast<double>(dimsB.x);
    //double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
    //                   (msecPerMatrixMul / 1000.0f);
    //printf(
    //    "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
    //    " WorkgroupSize= %u threads/block\n",
    //    gigaFlops,
    //    msecPerMatrixMul,
    //    flopsPerMatrixMul,
    //    threads.x * threads.y);

    //printf("CUDA= %.3f msec,", msecPerMatrixMul);

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    //printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero

    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], dimsA.x * valB, eps);
            correct = false;
        }
    }

    //printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");


    // Allocate CUDA events that we'll use for timing
    //hipEvent_t start2;
    //checkCudaErrors(hipEventCreate(&start2));

    //hipEvent_t stop2;
    //checkCudaErrors(hipEventCreate(&stop2));

    // Record the start event
    //checkCudaErrors(hipEventRecord(start2, NULL));
    
    auto start2 = high_resolution_clock::now();

    matrixMultiplyWithoutCUDA(h_C, h_A, h_B, dimsA, dimsB);
    
    auto stop2 = high_resolution_clock::now();

    
    auto duration2 = duration_cast<microseconds>(stop2 - start2);
    
    cout << duration2.count() << endl;


    // Record the stop event
    //checkCudaErrors(hipEventRecord(stop2, NULL));

    // Wait for the stop event to complete
    //checkCudaErrors(hipEventSynchronize(stop2));

    //float msc2 = 0.0f;
    //checkCudaErrors(hipEventElapsedTime(&msc2, start2, stop2));


    //printf("Without CUDA= %.3f msec\n", msc2);




    
    
    
    
    
    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    printf("\nNOTE: The CUDA Samples are not meant for performance"\
           "measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main(int argc, char **argv) {
    //printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices" \
               " must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    //int dev = findCudaDevice(argc, (const char **)argv); // This line prints the available CUDA compatible GPU information

    int matrix_result;

    int block_size = 32;
    
    bool alter = true;
    for (int x = 1, y = 1; x < 1024 && y < 1024; ) {
        
        dim3 dimsA(5 * 2 * block_size, x * block_size, 1);
        dim3 dimsB(y * block_size, 5 * 2 * block_size, 1);

        // width of Matrix A
        if (checkCmdLineFlag(argc, (const char**)argv, "wA")) {
            dimsA.x = getCmdLineArgumentInt(argc, (const char**)argv, "wA");
        }

        // height of Matrix A
        if (checkCmdLineFlag(argc, (const char**)argv, "hA")) {
            dimsA.y = getCmdLineArgumentInt(argc, (const char**)argv, "hA");
        }

        // width of Matrix B
        if (checkCmdLineFlag(argc, (const char**)argv, "wB")) {
            dimsB.x = getCmdLineArgumentInt(argc, (const char**)argv, "wB");
        }

        // height of Matrix B
        if (checkCmdLineFlag(argc, (const char**)argv, "hB")) {
            dimsB.y = getCmdLineArgumentInt(argc, (const char**)argv, "hB");
        }

        if (dimsA.x != dimsB.y) {
            printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
                dimsA.x, dimsB.y);
            exit(EXIT_FAILURE);
        }

        printf("MatrixA(%d,%d), MatrixB(%d,%d), ", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

        matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);



        if (alter) {
            x = x * 2;
        }
        else {
            y = y * 2;
        }
        alter = !alter;
    }

    

    //exit(matrix_result);
    return 3;
}

